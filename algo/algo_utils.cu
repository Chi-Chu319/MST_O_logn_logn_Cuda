#include "hip/hip_runtime.h"
#include "algo.cuh"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

__device__ int get_cluster_leader(int* cluster_ids, int v) {
    if (v < 0) {
        return v;
    }

    int leader = v;
    while (cluster_ids[leader] != leader) {
        leader = cluster_ids[leader];
    }

    return leader;
}

__device__ int get_cluster_machine(int num_vertex_local, int v) {
    return v / num_vertex_local;
}

__device__ void swap(ClusterEdge* a, ClusterEdge* b) {
    ClusterEdge temp = *a;
    *a = *b;
    *b = temp;
}

__device__ int partition(ClusterEdge* arr, int low, int high) {
    float pivot = arr[high].weight;
    int i = low - 1;
    for (int j = low; j < high; j++) {
        if (arr[j].weight <= pivot) {
            i++;
            swap(&arr[i], &arr[j]);
        }
    }
    swap(&arr[i + 1], &arr[high]);
    return i + 1;
}

__device__ void quickSortIterative(int* stack, ClusterEdge* arr, int l, int h) {
    int top = -1;

    stack[++top] = l;
    stack[++top] = h;

    while (top >= 0) {
        // Pop h and l
        h = stack[top--];
        l = stack[top--];

        int p = partition(arr, l, h);

        if (p - 1 > l) {
            stack[++top] = l;
            stack[++top] = p - 1;
        }

        if (p + 1 < h) {
            stack[++top] = p + 1;
            stack[++top] = h;
        }
    }
}

// removes all the empty slots in the array returns the size of the array after squeezing
__device__ int squeezeArray(ClusterEdge* edges, int n) {
    // points to the first non-empty slot after empty slots
    int ptr1 = 0;
    // points to the first empty slot
    int ptr2 = 0;

    while (ptr1 < n) {
        if (edges[ptr2].from_v != -1) {
            ptr2++;
            if (ptr1 < ptr2) {
                ptr1 = ptr2;
            }
        } else if (edges[ptr1].from_v == -1) {
            ptr1++;
        } else if (ptr2 < n && ptr1 < n) {
            edges[ptr2] = edges[ptr1];
            edges[ptr1] = ClusterEdge();
        }
    }

    return ptr2;
}

__global__ void min_to_cluster_kernel(ClusterEdge* to_cluster_buf, ClusterEdge* min_edges_buf, const float* vertices, int* cluster_ids, const int n, int num_vertices_local) {
    int i = threadIdx.x;
    int j = blockIdx.x;

    int vertex_local_start = (j * blockDim.x + i) * num_vertices_local;

    for (int vertex_local = 0; vertex_local < num_vertices_local; ++vertex_local) {
        int from_v = vertex_local + vertex_local_start;
        int min_edge_start = from_v * n;
        ClusterEdge* cluster_edges = min_edges_buf + min_edge_start;

        for (int k = 0; k < n; ++k) {
            cluster_edges[from_v * num_vertices_local + k] = ClusterEdge();
        }

        for (int to_v = 0; to_v < n; ++to_v) {
            float weight = vertices[from_v * n + to_v];
            int from_cluster = get_cluster_leader(cluster_ids, from_v);
            int to_cluster = get_cluster_leader(cluster_ids, to_v);

            if (from_cluster != to_cluster) {
                if (cluster_edges[to_cluster].from_v == -1) {
                    cluster_edges[to_cluster] = ClusterEdge(from_v, to_v, weight);
                } else if (weight < cluster_edges[to_cluster].weight) {
                    cluster_edges[to_cluster] = ClusterEdge(from_v, to_v, weight);
                }
            }
        }

        //  loop cluster_edges for non empty item and update cluster_ids
        for (int k = 0; k < n; ++k) {
            if (cluster_edges[k].from_v != -1) {
                ClusterEdge edge = cluster_edges[k];
                int to_cluster = get_cluster_leader(cluster_ids, edge.to_v);
                to_cluster_buf[edge.from_v * n + to_cluster] = edge;
            }
        }
    }
}

__global__ void min_from_cluster_kernel(const ClusterEdge* to_cluster_buf, ClusterEdge* from_cluster_buf, ClusterEdge* min_edges_bufGPU, int* min_edges_stack_bufGPU, int* cluster_ids, int* cluster_sizes, const int n, int num_vertices_local) {
    int vertex_local_start = (blockIdx.x * blockDim.x + threadIdx.x) * num_vertices_local;

    for (int vertex_local = 0; vertex_local < num_vertices_local; ++vertex_local) {
        int vertex = vertex_local + vertex_local_start;
        int min_edge_start = vertex * n;

        ClusterEdge* cluster_edges = min_edges_bufGPU + min_edge_start;
        int* stack = min_edges_stack_bufGPU + min_edge_start;

        if (get_cluster_leader(cluster_ids, vertex) != vertex) {
            continue;
        }

        for (int k = 0; k < n; ++k) {
            cluster_edges[k] = ClusterEdge();
        }

        for (int from_v = 0; from_v < n; ++from_v) {
            ClusterEdge edge = to_cluster_buf[from_v * n + vertex];

            if (edge.from_v != -1) {
                int from_cluster = get_cluster_leader(cluster_ids, edge.from_v);
                if (cluster_edges[from_cluster].from_v == -1) {
                    cluster_edges[from_cluster] = edge;
                } else if (edge.weight < cluster_edges[from_cluster].weight) {
                    cluster_edges[from_cluster] = edge;
                }
            }
        }

        int edge_size = 0;
        for (int k = 0; k < n; ++k) {
            if (cluster_edges[k].from_v != -1) {
                edge_size++;
            }
        }

        int squeezedSize = squeezeArray(cluster_edges, n);

        quickSortIterative(stack, cluster_edges, 0, squeezedSize - 1);

        int mu = min(cluster_sizes[vertex], edge_size);

        // For all non empty cluster_edges, update cluster_ids
        for (int k = 0; k < mu; ++k) {
            from_cluster_buf[vertex * n + k] = cluster_edges[k];
        }
    }
}

__global__ void speedup_kernel(const float* vertices, ClusterEdge* from_cluster_buf, const int n, int num_vertices_local) {
    int i = threadIdx.x;
    int j = blockIdx.x;

    int vertex_local_start = (j * blockDim.x + i) * num_vertices_local;

    for (int vertex_local = 0; vertex_local < num_vertices_local; ++vertex_local) {
        int from_v = vertex_local + vertex_local_start;

        float min_weight = DBL_MAX;
        int min_to_v;

        for (int to_v = 0; to_v < n; ++to_v) {
            if (from_v == to_v) {
                continue;
            }

            float weight = vertices[from_v * n + to_v];
            // update min_weight
            if (weight < min_weight) {
                min_weight = weight;
                min_to_v = to_v;
            }
        }

        from_cluster_buf[from_v * n + min_to_v] = ClusterEdge(from_v, min_to_v, min_weight);
    }
}

__global__ void min_to_cluster_kernel_sparse(ClusterEdge* to_cluster_buf, ClusterEdge* min_edges_buf, SparseGraphEdge* edges, int* v_indices, int* cluster_ids, const int n, int num_vertices_local) {
    int i = threadIdx.x;
    int j = blockIdx.x;

    int vertex_local_start = (j * blockDim.x + i) * num_vertices_local;

    for (int vertex_local = 0; vertex_local < num_vertices_local; ++vertex_local) {
        int from_v = vertex_local + vertex_local_start;

        ClusterEdge* cluster_edges = min_edges_buf + v_indices[from_v];
        SparseGraphEdge* edges_local = edges + v_indices[from_v];

        for (int k = 0; k < v_indices[from_v + 1] - v_indices[from_v]; ++k) {
            cluster_edges[k] = ClusterEdge();
        }

        int edge_count = 0;

        for (int i = 0; i < v_indices[from_v + 1] - v_indices[from_v]; ++i) {
            SparseGraphEdge edge = edges_local[i];
            float weight = edge.weight;
            int to_v = edge.to_v;

            int from_cluster = get_cluster_leader(cluster_ids, from_v);
            int to_cluster = get_cluster_leader(cluster_ids, to_v);

            if (from_cluster != to_cluster) {
                for (int k = 0; k < edge_count + 1; ++k) {
                    if (to_cluster == get_cluster_leader(cluster_ids, cluster_edges[k].to_v)) {
                        if (weight < cluster_edges[k].weight) {
                            cluster_edges[k] = ClusterEdge(from_v, to_v, weight);
                            edge_count++;
                        } 
                        break;
                    // empty slot
                    } else if (k == edge_count) {
                        cluster_edges[k] = ClusterEdge(from_v, to_v, weight);
                        edge_count++;
                        break;
                    }
                }
            }
        }

        //  loop cluster_edges for non empty item and update cluster_ids
        edge_count = 0;
        for (int k = 0; k < v_indices[from_v + 1] - v_indices[from_v]; ++k) {
            if (cluster_edges[k].from_v != -1) {
                ClusterEdge edge = cluster_edges[k];
                int to_cluster = get_cluster_leader(cluster_ids, edge.to_v);
                to_cluster_buf[v_indices[from_v] + edge_count] = edge;
                edge_count++;
            }
        }
    }
}

__global__ void min_from_cluster_kernel_sparse(
    const ClusterEdge* to_cluster_buf,
    ClusterEdge* from_cluster_buf,
    ClusterEdge* min_edges_bufGPU,
    int* min_edges_stack_bufGPU,
    int* v_indices,
    int* cluster_leader_sizesGPU,
    int* cluster_ids,
    int* cluster_sizes,
    const int n,
    int num_vertices_local
) {
    int vertex_local_start = (blockIdx.x * blockDim.x + threadIdx.x) * num_vertices_local;

    for (int vertex_local = 0; vertex_local < num_vertices_local; ++vertex_local) {
        int to_v = vertex_local + vertex_local_start;

        if (get_cluster_leader(cluster_ids, to_v) != to_v) {
            continue;
        }

        ClusterEdge* cluster_edges = min_edges_bufGPU + cluster_leader_sizesGPU[to_v];
        int* stack = min_edges_stack_bufGPU + cluster_leader_sizesGPU[to_v];

        for (int k = 0; k < cluster_leader_sizesGPU[to_v + 1] - cluster_leader_sizesGPU[to_v]; ++k) {
            cluster_edges[k] = ClusterEdge();
        }

        int edge_count = 0;

        for (int from_v = 0; from_v < n; ++from_v) {
            ClusterEdge edge = ClusterEdge();

            //  the most inefficient part for sparse graph
            for (int i = v_indices[from_v]; i < v_indices[from_v + 1]; ++i) {
                ClusterEdge e = to_cluster_buf[i];
                if (get_cluster_leader(cluster_ids, e.to_v) == to_v) {
                    edge = e;
                    break;
                }
            }

            if (edge.from_v != -1) {
                for (int k = 0; k < edge_count + 1; ++k) {
                    if (get_cluster_leader(cluster_ids, edge.from_v) == get_cluster_leader(cluster_ids, cluster_edges[k].from_v)) {
                        if (edge.weight < cluster_edges[k].weight) {
                            cluster_edges[k] = edge;
                            edge_count++;
                        } 
                        break;
                    // empty slot
                    } else if (k == edge_count) {
                        cluster_edges[k] = edge;
                        edge_count++;
                        break;
                    }
                }
            }
        }

        quickSortIterative(stack, cluster_edges, 0, edge_count - 1);

        int mu = min(cluster_sizes[to_v], edge_count);

        // For all non empty cluster_edges, update cluster_ids
        for (int k = 0; k < mu; ++k) {
            from_cluster_buf[cluster_leader_sizesGPU[to_v] + k] = cluster_edges[k];
        }
    }
}