#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "algo.cuh"
#include <float.h>

int main() {
    int n_block = 1;
    int n_thread = 4;
    const int n = 8192;
    int num_vertex_local = 2048;

    srand(time(0)); 

    double vertices[n * n];
    for (int i = 0; i < n; ++i) {
        for (int j = i + 1; j < n; ++j) {
            vertices[i * n + j] = (double)rand() / RAND_MAX;
            vertices[j * n + i] = vertices[i * n + j];
        }
    }

    for (int i = 0; i < n; ++i) {
        vertices[i * n + i] = 200;
    }

    // start timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


    std::vector<ClusterEdge> cuda_result = MSTSolver::algo_cuda(vertices, n, n_block, n_thread, num_vertex_local);
    // std::vector<int> prim_parents = MSTSolver::algo_prim(vertices, n);

    // end timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time: %f\n", milliseconds);


    // double cuda_weights = 0;
    // for (int i = 0; i < cuda_result.size(); i++) {
    //     cuda_weights += cuda_result[i].weight;
    // }

    // double prim_weights = 0;
    // for (int i = 1; i < n; i++) {
    //     prim_weights += vertices[i * n + prim_parents[i]];
    // }

    // printf("CUDA: %f\n", cuda_weights);
    // printf("Prim: %f\n", prim_weights);
}
