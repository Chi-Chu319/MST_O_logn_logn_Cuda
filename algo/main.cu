#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "algo.cuh"
#include <float.h>

int main() {
    int n_block = 4;
    int n_thread = 1024;
    const int n = 4096;

    double vertices[n * n];
    for (int i = 0; i < n; ++i) {
        for (int j = i + 1; j < n; ++j) {
            vertices[i * n + j] = (double)rand() / RAND_MAX;
            vertices[j * n + i] = vertices[i * n + j];
        }
    }

    for (int i = 0; i < n; ++i) {
        vertices[i * n + i] = 200;
    }

    std::vector<ClusterEdge> cuda_result = MSTSolver::algo_cuda(vertices, n, n_block, n_thread);
    std::vector<int> prim_parents = MSTSolver::algo_prim(vertices, n);

    double cuda_weights = 0;
    for (int i = 0; i < cuda_result.size(); i++) {
        cuda_weights += cuda_result[i].weight;
    }

    double prim_weights = 0;
    for (int i = 1; i < n; i++) {
        prim_weights += vertices[i * n + prim_parents[i]];
    }

    printf("CUDA: %f\n", cuda_weights);
    printf("Prim: %f\n", prim_weights);
}
