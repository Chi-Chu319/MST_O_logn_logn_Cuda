#include "hip/hip_runtime.h"
#include "algo.cuh"
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <map>
#include <algorithm>
#include <set>

static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

int get_cluster_leader_host(int* cluster_ids, int v) {
    int leader = v;
    while (cluster_ids[leader] != leader) {
        leader = cluster_ids[leader];
    }

    return leader;
}

bool get_cluster_finished(int* cluster_ids, bool* cluster_finished, int v) {
    int leader = get_cluster_leader_host(cluster_ids, v);
    return cluster_finished[leader];
}

bool cluster_safe_union(int* cluster_ids, int* cluster_size, int p, int q) {
    int i = get_cluster_leader_host(cluster_ids, p);
    int j = get_cluster_leader_host(cluster_ids, q);

    if (i == j) {
        return false;
    }

    if (cluster_size[i] < cluster_size[j]) {
        cluster_ids[i] = j;
        cluster_size[j] += cluster_size[i];
        return true;
    } else {
        cluster_ids[j] = i;
        cluster_size[i] += cluster_size[j];
        return true;
    }
}

void cluster_set_finished(int* cluster_ids, bool* cluster_finished, int i) {
    cluster_finished[get_cluster_leader_host(cluster_ids, i)] = true;
}

namespace MSTSolver {

    //  n is the number of vertices
    std::vector<ClusterEdge> algo_cuda(const double* vertices, int n, int n_block, int n_thread, int num_vertices_local) {
        float cpu_time = 0;

        int num_vertices = n;
        int k = 0;

        int* cluster_ids = new int[n];

        for (int i = 0; i < n; ++i) {
            cluster_ids[i] = i;
        }

        int* cluster_sizes = new int[n];

        for (int i = 0; i < n; ++i) {
            cluster_sizes[i] = 1;
        }

        std::vector<ClusterEdge> mst_edges = std::vector<ClusterEdge>();
        int* cluster_idsGPU = NULL;
        CHECK(hipMalloc((void**)&cluster_idsGPU, n * sizeof(int)));
        CHECK(hipMemcpy(cluster_idsGPU, cluster_ids, n * sizeof(int), hipMemcpyHostToDevice));
        
        double* verticesGPU = NULL;
        CHECK(hipMalloc((void**)&verticesGPU, n * n * sizeof(double)));
        CHECK(hipMemcpy(verticesGPU, vertices, n * n * sizeof(double), hipMemcpyHostToDevice));
        
        ClusterEdge* to_cluster_bufGPU = NULL;
        CHECK(hipMalloc((void**)&to_cluster_bufGPU, n * n * sizeof(ClusterEdge)));

        ClusterEdge* from_cluster_bufGPU = NULL;
        CHECK(hipMalloc((void**)&from_cluster_bufGPU, n * n * sizeof(ClusterEdge)));

        int num_clusters = num_vertices;

        int *deviceResult; // Device pointer to memory

        // Allocate device memory for the result
        hipMalloc((void **)&deviceResult, sizeof(int));

        ClusterEdge* from_cluster_buf = new ClusterEdge[n * n];

        while (true) {
            CHECK(hipMemset(to_cluster_bufGPU, 0, n * n * sizeof(ClusterEdge)));
            CHECK(hipMemset(from_cluster_bufGPU, 0, n * n * sizeof(ClusterEdge)));
            CHECK(hipMemcpy(cluster_idsGPU, cluster_ids, n * sizeof(int), hipMemcpyHostToDevice));


            // if (k == 0) {
            //     speedup_kernel<<<n_block, n_thread>>>(vertices, from_cluster_buf, n, num_vertices_local);
            //     CHECK(hipGetLastError());
            // }
            // else {
                min_to_cluster_kernel<<<n_block, n_thread>>>(to_cluster_bufGPU, verticesGPU, cluster_idsGPU, n, num_vertices_local, k);
                CHECK(hipGetLastError());

                min_from_cluster_kernel<<<n_block, n_thread>>>(to_cluster_bufGPU, from_cluster_bufGPU, cluster_idsGPU, n, num_vertices_local);
                CHECK(hipGetLastError());
            // }

            CHECK(hipDeviceSynchronize());

            CHECK(hipMemcpy(from_cluster_buf, from_cluster_bufGPU, n * n * sizeof(ClusterEdge), hipMemcpyDeviceToHost));

            // start timer
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
            hipEventSynchronize(start);

            // rank 0 merge edges
            std::vector<ClusterEdge> edges_to_add;

            for (int i = 0; i < n; ++i) {
                for (int j = 0; j < n; ++j) {
                    if (from_cluster_buf[i * n + j].from_v != 0) {
                        edges_to_add.push_back(from_cluster_buf[i * n + j]);
                    }
                }
            }

            std::sort(edges_to_add.begin(), edges_to_add.end(), [](ClusterEdge a, ClusterEdge b) {
                return a.weight < b.weight;
            });

            std::vector<bool> heaviest_edges(edges_to_add.size());
            std::fill(heaviest_edges.begin(), heaviest_edges.end(), false);
            std::map<int, bool> encountered_clusters;

            for (int i = edges_to_add.size() - 1; i >= 0; --i) {
                ClusterEdge edge = edges_to_add[i];
                int to_cluster = get_cluster_leader_host(cluster_ids, edge.to_v);

                if (encountered_clusters.find(to_cluster) == encountered_clusters.end()) {
                    encountered_clusters[to_cluster] = true;
                    heaviest_edges[i] = true;
                }
            }

            // declare a bool array with size n and fill it with false
            bool* cluster_finished = new bool[n];
            std::fill(cluster_finished, cluster_finished + n, false);

            for (int i = 0; i < edges_to_add.size(); ++i) {
                ClusterEdge edge = edges_to_add[i];
                int from_cluster = get_cluster_leader_host(cluster_ids, edge.from_v);
                int to_cluster = get_cluster_leader_host(cluster_ids, edge.to_v);

                bool from_cluster_finished = get_cluster_finished(cluster_ids, cluster_finished, from_cluster);
                bool to_cluster_finished = get_cluster_finished(cluster_ids, cluster_finished, to_cluster);
                
                if (to_cluster_finished && from_cluster_finished) {
                    continue;
                }

                bool merged = cluster_safe_union(cluster_ids, cluster_sizes, from_cluster, to_cluster);

                if (merged) {
                    mst_edges.push_back(edge);
                    if (heaviest_edges[i] || (from_cluster_finished || to_cluster_finished)) {
                        cluster_set_finished(cluster_ids, cluster_finished, from_cluster);
                        cluster_set_finished(cluster_ids, cluster_finished, to_cluster);
                    }
                } else {
                    if (heaviest_edges[i]) {
                        cluster_set_finished(cluster_ids, cluster_finished, to_cluster);
                    }
                }
            }

            delete[] cluster_finished;

            // count the number of unique numbers in cluster_ids
            std::set<int> unique_cluster_finder_id(cluster_ids, cluster_ids + num_vertices);
            num_clusters = unique_cluster_finder_id.size();

            k++;

            if (k >= 10) {
                throw std::runtime_error("k >= 10");
            }

            if (num_clusters == 1) {
                break;
            }

            // flatten cluster_ids
            int* new_cluster_ids = new int[n];
            for (int i = 0; i < n; i++) {
                new_cluster_ids[i] = get_cluster_leader_host(cluster_ids, i);
            }

            delete[] cluster_ids;
            cluster_ids = new_cluster_ids;

            for (int i = 0; i < n; ++i) {
                cluster_sizes[i] = 1;
            }

            // end timer
            hipEventRecord(stop);
            hipEventSynchronize(stop);

            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            cpu_time += milliseconds;
        }

        CHECK(hipFree(verticesGPU));
        CHECK(hipFree(cluster_idsGPU));
        CHECK(hipFree(to_cluster_bufGPU));
        CHECK(hipFree(from_cluster_bufGPU));


        delete[] cluster_ids;
        delete[] cluster_sizes;
        delete[] from_cluster_buf;

        // print cpu time
        std::cout << "CPU time: " << cpu_time << std::endl;

        return mst_edges;
    }

    std::vector<int> algo_prim(const double* vertices, const int n) {
        std::vector<int> parent(n, -1);
        std::vector<double> key(n, std::numeric_limits<double>::max());
        std::vector<bool> mstSet(n, false);

        key[0] = 0;
        for (int count = 0; count < n - 1; count++) {
            int u = -1;
            double min_key = std::numeric_limits<double>::max();
            for (int i = 0; i < n; i++) {
                if (!mstSet[i] && key[i] < min_key) {
                    u = i;
                    min_key = key[i];
                }
            }
            mstSet[u] = true;
            for (int v = 0; v < n; v++) {
                if (!mstSet[v] && vertices[u * n + v] < key[v]) {
                    parent[v] = u;
                    key[v] = vertices[u * n + v];
                }
            }
        }

        return parent;
    }

}

